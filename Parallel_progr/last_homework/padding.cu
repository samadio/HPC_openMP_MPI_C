
#include <hip/hip_runtime.h>
#include <stdio.h>
#include<sys/time.h>
#include<math.h>

#define N 8192
#define nth 1024

__global__ void fast_transpose(size_t* A, size_t* B){
    __shared__ size_t Ablock[nth];
    __shared__ size_t Bblock[nth];

    size_t dimx=blockDim.x;
    size_t dimy=blockDim.y;
    
    //dimx=linear dimension in x of a submatrix block
    size_t th=threadIdx.x+threadIdx.y*dimx;
    size_t thx=threadIdx.x;
    size_t thy=threadIdx.y;

    size_t starty=blockIdx.y*N*dimy;
    size_t startx=blockIdx.x*dimx;
    size_t start= startx+starty;
    //Ablock is different for every block, so I can index it with th
    Ablock[th]= A[start+thx+(thy)*(N)];
    //creation of A completed for each block
    __syncthreads();

    //transpose into B block
    Bblock[dimy*thx + thy] = Ablock[th];

    __syncthreads();


    //put Bblock in B
    start=blockIdx.y*dimy+dimx*N*blockIdx.x; //the x block index of the original matrix becomes y index of transpose, so skip N
    B[ start+thy+(thx)*(N) ]=Bblock[dimy*thx + thy];

}

__global__ void transpose(size_t* A, size_t *B){
  size_t j=blockIdx.x;
  size_t i=threadIdx.x;
  while(i<N){
    B[j+i*N]=A[i+j*N];
    i+=blockDim.x;
  }
  
}

                                        /////////////////////C utilites//////////////////////////////

int transposed(size_t *A, size_t* At){
  size_t i,j;
   for(i=0;i<N;i++){
     for(j=0;j<N;j++){
       if(A[i+j*N]!=At[j+i*N]){return 0;}
     }
   }
   return 1;
}

double seconds()

{

  struct timeval tmp;
  double sec;
  gettimeofday( &tmp, (struct timezone *)0 );
  sec = tmp.tv_sec + ((double)tmp.tv_usec)/1000000.0;
  return sec;

}

                            ////////////////////////////////////main
int main(){
  size_t elements=N*N;
  size_t space=N*N*sizeof(size_t);

  size_t*A=(size_t*)malloc(space);
  size_t*dev_A;
  size_t*B=(size_t*)malloc(space);
  size_t*dev_B;

  size_t i;
  for(i=0;i<elements;i++){
      A[i]=i%N;
  }

  hipMalloc( (void**)&dev_A, space );
  hipMalloc( (void**)&dev_B, space );

  hipMemcpy( dev_A, A, space, hipMemcpyHostToDevice );

  double tstart=seconds();
  transpose<<< N, nth >>>(dev_A, dev_B); 
  hipDeviceSynchronize();
  double duration=seconds()-tstart;
  printf("transp time: %lf\n",duration);

  hipMemcpy( B, dev_B, space, hipMemcpyDeviceToHost );

  printf("correct? %d\n\n",transposed(A,B));

  size_t block_side= (size_t)sqrt(nth);
  dim3 grid,block;
  if(block_side*block_side==nth){
    grid.x=grid.y=N/block_side;  //number of orizontal blocks=number of vertical blocks
    block.x=block.y=block_side;  //block linear length
  }
  else{
    grid.x=N/32; //ideally, we should have an algorithm that given nth finds (a,b) integers such that nth=a*b and (a,b) closest to each other
    grid.y=N/16; //to be preferred a>b, so that we read more often on x (continous in memory)
    block.x=32;
    block.y=16;
  }
  
  tstart=seconds();
  fast_transpose<<< grid, block >>>(dev_A, dev_B);
  hipDeviceSynchronize();
  duration=seconds()-tstart;
  printf("fast times: %lf\n",duration);

  hipMemcpy( B, dev_B, space, hipMemcpyDeviceToHost );

/*  for(i=0;i<elements;i++){
     if(i%N==0 && i!=0)printf("\n");  
     printf("%d ", A[i]);
  }
  printf("\n");
               
               
  for(i=0;i<elements;i++){
    if(i%N==0 && i!=0)printf("\n");
    
    printf("%d ", B[i]);
    }
  printf("\n"); */

  printf("correct? %d\n\n",transposed(A,B));

  free(A);free(B);
  hipFree(dev_A);hipFree(dev_B);
}
