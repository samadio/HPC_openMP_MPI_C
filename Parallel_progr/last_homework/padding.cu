
#include <hip/hip_runtime.h>
#include <stdio.h>
#include<math.h>
#define N 8192
#define nth 1024

__global__ void fast_transpose(size_t* A, size_t* B,size_t dim){
    __shared__ size_t Ablock[nth];
    __shared__ size_t Bblock[nth];

    //dim=linear dimension of a submatrix block
    size_t th=threadIdx.x+threadIdx.y*dim;
    size_t thx=threadIdx.x;
    size_t thy=threadIdx.y;

    size_t starty=blockIdx.y*N*dim;
    size_t startx=blockIdx.x*dim;
    size_t start= startx+starty;
    //Ablock is different for every block, so I can index it with th
    Ablock[th]= A[start+thx+(thy)*(N)];
    //creation of A completed for each block
    __syncthreads();

    //transpose into B block
    Bblock[dim*thx + thy] = Ablock[th];

    __syncthreads();


    //put Bblock in B
    start=blockIdx.y*dim+dim*N*blockIdx.x; //the x block index of the original matrix becomes y index of transpose, so skip N
    B[ start+thy+(thx)*(N) ]=Bblock[dim*thx + thy];

}

int transposed(size_t *A, size_t* At){
  size_t i,j;
   for(i=0;i<N;i++){
     for(j=0;j<N;j++){
       if(A[i+j*N]!=At[j+i*N]){return 0;}
     }
   }
   return 1;
}


int main(){
    size_t elements=N*N;
    size_t space=N*N*sizeof(size_t);

    size_t*A=(size_t*)malloc(space);
    size_t*dev_A;
    size_t*B=(size_t*)malloc(space);
    size_t*dev_B;

    size_t i;
    for(i=0;i<elements;i++){
        A[i]=i%N;
    }

    hipMalloc( (void**)&dev_A, space );
    hipMalloc( (void**)&dev_B, space );

    hipMemcpy( dev_A, A, space, hipMemcpyHostToDevice );

  size_t block_side= (size_t)sqrt(nth);
  dim3 grid,block;
  grid.x=grid.y=N/block_side;  //number of orizontal blocks=number of vertical blocks
  block.x=block.y=block_side;  //block linear length
  
  fast_transpose<<< grid, block >>>(dev_A, dev_B,block_side);
  
  hipMemcpy( B, dev_B, space, hipMemcpyDeviceToHost );

/*  for(i=0;i<elements;i++){
     if(i%N==0 && i!=0)printf("\n");  
     printf("%d ", A[i]);
  }
  printf("\n");*/
               
               
  printf("%d\n",transposed(A,B));
  for(i=0;i<elements;i++){
    if(i%N==0 && i!=0)printf("\n");
    
    printf("%d ", B[i]);
    }
  printf("\n"); 

  free(A);free(B);
  hipFree(dev_A);hipFree(dev_B);
}
