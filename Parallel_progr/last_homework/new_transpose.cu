#include "hip/hip_runtime.h"
#include <stdio.h>
#include<math.h>
#define N 80
#define nth 4

__global__ void fast_transpose(size_t* A, size_t* B,size_t dim){
    __shared__ Ablock[nth];
    __shared__ Bblock[nth]

    size_t length=blockDim.x; //dim of a single block: (nth)
    
    size_t th=threadIdx.x+threadIdx.y*dim;
    size_t thx=threadIdx.x;
    size_t thy=threadIdx.y;

    size_t starty=blockIdx.y*N;
    size_t startx=blockIdx.x*dim;
    size_t start= startx+starty;
    //Ablock is different for every block, so I can index it with th
    //go on until nth==dim, then skip N numbers
    Ablock[th]= A[start+thx+(thy)*(N)];
    //creation of A completed for each block
    __syncthreads();

    //transpose B
    Bblock[dim*thy + thx] = Ablock[th];

    __syncthreads();


    //put Bblock back in B
    start=blockIdx.y*dim+N*blockIdx.x; //the x block index of the original matrix becomes y index of transpose, so skip N
    
    B[ start+thx+(thy)*(N) ]=Bblock[dim*thy + thx];

}


in main(){
    size_t elements=80*80;
    size_t space=80*80*sizeof(size_t);

    size_t*A=(size_t*)malloc(space);
    size_t*dev_A;
    size_t*B=(size_t*)malloc(space);
    size_t*dev_B;

    size_t i;
    for(i=0;i<elements;i++){
        A[i]=i%N;
    }

    hipMalloc( (void**)&dev_A, space );
    hipMalloc( (void**)&dev_B, space );

    hipMemcpy( dev_A, A, space, hipMemcpyHostToDevice );

  size_t block_side= (size_t)sqrt(nth);
  dim3 grid,block;
  grid.x=grid.y=N/block_side;  //number of orizontal blocks=number of vertical blocks
  block.x=block.y=block_side;  //block linear length
  
  fast_transpose<<< grid, block >>>(dev_tableA, dev_tableB,block_side);
  

    free(A);free(B);
    hipFree(dev_A);cudafree(dev_B);
}