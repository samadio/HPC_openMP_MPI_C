#include "hip/hip_runtime.h"
#include <stdio.h>
#include<math.h>

#define row 8192
#define col 8192
#define space 8192*8192*sizeof(size_t)
#define elements 8192*8192

#define th_per_block 1024

///////////////////CUDA///////////////////
__global__ void initialize_table (size_t* A, size_t** table, size_t ncol){
  size_t i=threadIdx.x;
  size_t j=blockIdx.x;
  table[i]=A+j*gridDim.x+i*ncol;
}


__global__ void transpose (size_t** A, size_t** B,size_t cols){
  size_t i=blockIdx.x;
  size_t j=threadIdx.x;
  while(i<cols){
    B[j][i]=A[i][j];
  i+=blockDim.x;
}

}

__global__ void fast_transpose (size_t** tableA, size_t** tableB, const size_t dim){
  __shared__ size_t miniblockA[th_per_block];
  __shared__ size_t miniblockB[th_per_block];
  if(threadIdx.x==0 && threadIdx.y==0){
    size_t i;
    for(i=0;i<th_per_block;i++){
      miniblockA[i]=tableA[dim*blockIdx.y+i/dim][dim*blockIdx.x+i%dim];
    }
  }
  __syncthreads();

  miniblockB[dim*threadIdx.y+threadIdx.x]=miniblockA[dim*threadIdx.x+threadIdx.y];

  __syncthreads();

  if(threadIdx.x==0 && threadIdx.y==0){
    size_t i;
    for(i=0;i<th_per_block;i++){
      tableB[dim*blockIdx.x+i/dim][dim*blockIdx.y+i%dim]=miniblockB[i];
    }
  }

}

//////////////////////C utilites

void print_is_transpose(size_t *mat, size_t *transp, const size_t n){
    
  size_t i, j;
  for (i = 0; i < n; ++i){
for (j = 0; j < n; ++j)
    printf("%d ",(mat[i*n + j] != transp[j*n + i]) ? 0 : 1);
putchar('\n');
  }
}

int main() {

  size_t*A=(size_t*)malloc(space);
  size_t* dev_A;
  size_t* B=(size_t*)malloc(space);
  size_t* dev_B;
  
  size_t i;
  for(i=0;i<elements;i++){
    A[i]=i%row;
  }
  
  size_t** dev_tableA;
  size_t** dev_tableB;
  
  hipMalloc( (void**)&dev_A, space );
  hipMalloc( (void**)&dev_B, space );
  hipMalloc( (void***)&dev_tableA, row*sizeof(size_t) );
  hipMalloc( (void***)&dev_tableB, col*sizeof(size_t) );
  
  hipMemcpy( dev_A, A, space, hipMemcpyHostToDevice ); //send data to device
  
  if(row<=1024 && col<=1024){
    initialize_table<<< 1, row >>>(dev_A, dev_tableA,col);
    initialize_table<<< 1, col >>>(dev_B, dev_tableB,row);  
  }
  else{
    initialize_table<<< elements/th_per_block, th_per_block >>>(dev_A, dev_tableA,col);
    initialize_table<<< elements/th_per_block, th_per_block >>>(dev_B, dev_tableB,row);
  }
  
  // launch transpose() kernel
  transpose<<< elements/th_per_block, th_per_block >>>(dev_tableA, dev_tableB,col);
  
  size_t dim= (size_t)sqrt(th_per_block);
  dim3 grid,block;
  grid.x=col/dim;
  grid.y=row/dim;
  block.x=dim;
  block.y=dim;
  
  fast_transpose<<< grid, block >>>(dev_tableA, dev_tableB,dim);
  
  // copy device result back to host copy of c
  hipMemcpy( B, dev_B, space, hipMemcpyDeviceToHost );
  
/*  for(i=0;i<elements;i++){
  if(i%col==0 && i!=0)printf("\n");
  printf("%d ", A[i]);
  }
  printf("\n"); */
  
  for(i=0;i<elements;i++){
  if(i%row==0 && i!=0)printf("\n");
  
  printf("%d ", B[i]);
  }
  printf("\n");
 

<<<<<<< HEAD
  print_is_transpose(A,B, col); 
=======
  //print_is_transpose(mat_array, transp_array, N); 
>>>>>>> 9a442a36d2458910f4b2d0f9c80aaae55e549740
  free(A); free(B);
  hipFree( dev_A ); hipFree( dev_B ); hipFree(dev_tableA);hipFree(dev_tableB);
  return 0;
  }
