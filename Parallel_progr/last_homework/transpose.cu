#include "hip/hip_runtime.h"
#include <stdio.h>
#include<math.h>
#define row 8192
#define col 8192
#define space 8192*8192*sizeof(int)
#define elements 8192

#define th_per_block 64


//this works until col<=Nthreads. In this case every threads moves only 1 data

__global__ void initialize_table (int* A, int** table, int ncol){
  int i=threadIdx.x;
  table[i]=A+i*ncol;  
}


__global__ void transpose (int** A, int** B,size_t cols){
  size_t i=blockIdx.x;
  size_t j=threadIdx.x;
  while(i<cols){
    B[j][i]=A[i][j];
    i+=blockDim.x;
  }

}

__global__ void fast_transpose (int** tableA, int** tableB, const size_t dim){
  __shared__ int miniblockA[blockDim.x];
  __shared__ int miniblockB[blockDim.x];
  size_t dim= (size_t) sqrt(size);
  if(threadIdx.x==0 && threadIdx.y==0){
    size_t i;  
    for(i=0;i<blockDim.x;i++){
        miniblockA[i]=tableA[dim*blockIdx.y+i/dim][dim*blockIdx.x+i%dim];
      }
  }
  __sychthreads();

  miniblockB[dim*threadIdx.x+threadIdx.x]=miniblockA[dim*threadIdx.x+threadIdx.y];

  __sychthreads();

  if(threadIdx.x==0 && threadIdx.y==0){
    for(i=0;i<blockDim.x;i++){
      tableB[dim*blockIdx.y+i/dim][dim*blockIdx.x+i%dim]=miniblockB[i];
    }
  }

}


int main() {
    
  int*A=(int*)malloc(space);
  int* dev_A;
  int*B=(int*)malloc(space);
  int* dev_B;

  int i;
  for(i=0;i<elements;i++){
    A[i]=i;
  }

  int** dev_tableA;
  int** dev_tableB;

  // allocate device copies of A and B
  hipMalloc( (void**)&dev_A, space );
  hipMalloc( (void**)&dev_B, space );
  hipMalloc( (void***)&dev_tableA, row*sizeof(int) );
  hipMalloc( (void***)&dev_tableB, col*sizeof(int) );

  hipMemcpy( dev_A, A, space, hipMemcpyHostToDevice ); //send data to device

  initialize_table<<< 1, row >>>(dev_A, dev_tableA,col); 
  initialize_table<<< 1, col >>>(dev_B, dev_tableB,row); 


  // launch transpose() kernel
  transpose<<< elements/th_per_block, th_per_block >>>(dev_tableA, dev_tableB,col,dim); 

  int dim= (int)sqrt(th_per_block);
  dim3 grid,block;
  grid.x=col/dim;
  grid.y=row/dim;
  block.x=dim;
  block.y=dim;

  fast_transpose<<< row*col/th_per_block, th_per_block >>>(dev_tableA, dev_tableB,dim); 

  // copy device result back to host copy of c
  hipMemcpy( B, dev_B, space,   hipMemcpyDeviceToHost );

  for(i=0;i<elements;i++){
    if(i%col==0 && i!=0)printf("\n");
      printf("%d ", A[i]);
  }
  printf("\n");

  for(i=0;i<elements;i++){
    if(i%row==0 && i!=0)printf("\n");
    printf("%d ", B[i]);
  }
  printf("\n");

  free(A); free(B);
  hipFree( dev_A ); hipFree( dev_B ); hipFree(tableA);hipFree(tableB);
  return 0;
}
