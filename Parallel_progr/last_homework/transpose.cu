
#include <hip/hip_runtime.h>
#include <stdio.h>
#include<math.h>

#define row 64
#define col 64
#define space 64*64*sizeof(size_t)
#define elements 64*64

#define th_per_block 16


__global__ void initialize_table (size_t* A, size_t** table, size_t ncol){
  size_t i=threadIdx.x;
  table[i]=A+i*ncol;
}


__global__ void transpose (size_t** A, size_t** B,size_t cols){
  size_t i=blockIdx.x;
  size_t j=threadIdx.x;
  while(i<cols){
    B[j][i]=A[i][j];
  i+=blockDim.x;
}

}

__global__ void fast_transpose (size_t** tableA, size_t** tableB, const size_t dim){
  __shared__ size_t miniblockA[th_per_block];
  __shared__ size_t miniblockB[th_per_block];
  if(threadIdx.x==0 && threadIdx.y==0){
    size_t i;
    for(i=0;i<th_per_block;i++){
      miniblockA[i]=tableA[dim*blockIdx.y+i/dim][dim*blockIdx.x+i%dim];
    }
  }
  __syncthreads();

  miniblockB[dim*threadIdx.y+threadIdx.x]=miniblockA[dim*threadIdx.x+threadIdx.y];

  __syncthreads();

  if(threadIdx.x==0 && threadIdx.y==0){
    size_t i;
    for(i=0;i<th_per_block;i++){
      tableB[dim*blockIdx.x+i/dim][dim*blockIdx.y+i%dim]=miniblockB[i];
    }
  }

}




int main() {

  size_t*A=(size_t*)malloc(space);
  size_t* dev_A;
  size_t* B=(size_t*)malloc(space);
  size_t* dev_B;
  
  size_t i;
  for(i=0;i<elements;i++){
    A[i]=i;
  }
  
  size_t** dev_tableA;
  size_t** dev_tableB;
  
  hipMalloc( (void**)&dev_A, space );
  hipMalloc( (void**)&dev_B, space );
  hipMalloc( (void***)&dev_tableA, row*sizeof(size_t) );
  hipMalloc( (void***)&dev_tableB, col*sizeof(size_t) );
  
  hipMemcpy( dev_A, A, space, hipMemcpyHostToDevice ); //send data to device
  
  initialize_table<<< 1, row >>>(dev_A, dev_tableA,col);
  initialize_table<<< 1, col >>>(dev_B, dev_tableB,row);
  
  
  // launch transpose() kernel
  transpose<<< elements/th_per_block, th_per_block >>>(dev_tableA, dev_tableB,col);
  
  size_t dim= (size_t)sqrt(th_per_block);
  dim3 grid,block;
  grid.x=col/dim;
  grid.y=row/dim;
  block.x=dim;
  block.y=dim;
  
  fast_transpose<<< grid, block >>>(dev_tableA, dev_tableB,dim);
  
  // copy device result back to host copy of c
  hipMemcpy( B, dev_B, space, hipMemcpyDeviceToHost );
  
  for(i=0;i<elements;i++){
  if(i%col==0 && i!=0)printf("\n");
  printf("%d ", A[i]);
  }
  printf("\n");
  
  for(i=0;i<elements;i++){
  if(i%row==0 && i!=0)printf("\n");
  
  printf("%d ", B[i]);
  }
  printf("\n");
  
  free(A); free(B);
  hipFree( dev_A ); hipFree( dev_B ); hipFree(dev_tableA);hipFree(dev_tableB);
  return 0;
  }
