
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N  512

__global__ void add( int *a, int *b, int *c ) {
       c[blockIdx.x] = a[blockIdx.x]+b[blockIdx.x];
}

void random_ints(int *p, int n) {
	int i;
	for(i=0; i<n; i++) {
		p[i]=rand();
	}
}

int main( void ) {
    int *a, *b, *c, *d;               // host copies of a, b, c
    int *dev_a, *dev_b, *dev_c;   // device copies of a, b, c
    int size = N * sizeof( int ); // we need space for 512   									// integers
    int i;

    // allocate device copies of a, b, c
    hipMalloc( (void**)&dev_a, size );
    hipMalloc( (void**)&dev_b, size );
    hipMalloc( (void**)&dev_c, size );

    a = (int*)malloc( size ); 
    b = (int*)malloc( size );
    c = (int*)malloc( size );
    d = (int*)malloc( size );

    random_ints( a, N ); 
    random_ints( b, N );
    // copy inputs to device
   hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice );
   hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice );

    // launch add() kernel with N parallel blocks
    add<<< N, 1 >>>( dev_a, dev_b, dev_c );

    // copy device result back to host copy of c
   hipMemcpy( c, dev_c, size,   hipMemcpyDeviceToHost );

    for(i=0; i<N; i++) {
	d[i]=a[i]+b[i];
	if(d[i]!=c[i]) {
	   printf("error: expected %d, got %d!\n",c[i], d[i]);
	   break;
	}
    }
    if(i==N) {
	   printf("correct!\n");
    }    

 
    free( a ); free( b ); free( c );
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );
    return 0;
}
