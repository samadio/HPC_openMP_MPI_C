#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ reverse (int* in, int* out){
    out[blockDim.x-threadIdx.x]=in[threadIdx.x];
}


int main() {
    int d_in={100,110,200,220,300};
    int size = 5* sizeof( int ); // we need space for 5 integers
    int* d_out=(int*)malloc(size);
    int *dev_in, *dev_out,  // device copies 
    
    // allocate device copies of dev_in/out
    hipMalloc( &dev_in, size );
    hipMalloc( &dev_out, size );

   hipMemcpy( dev_in, d_in, size, hipMemcpyHostToDevice ); //send data to device

    // launch reverse() kernel
    reverse<<< 1, size >>>(dev_in, dev_out); //1 block of size threads

    // copy device result back to host copy of c
   hipMemcpy( d_out, dev_out, size,   hipMemcpyDeviceToHost );

    free( d_in );
    free( d_out );

    hipFree( dev_in );
    hipFree( dev_out );
    return 0;
}