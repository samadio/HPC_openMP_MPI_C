
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void reverse (int* in, int* out){
    out[blockDim.x-threadIdx.x-1]=in[threadIdx.x];
}


int main() {
    int d_in[]={100,110,200,220,300};
    int size = 5* sizeof( int );
    int* d_out=(int*)malloc(size);
    int *dev_in, *dev_out;  // device copies 
    int i;
    
    // allocate device copies of dev_in/out
    hipMalloc( (void**)&dev_in, size );
    hipMalloc( (void**)&dev_out, size);

   hipMemcpy( dev_in, d_in, size, hipMemcpyHostToDevice ); //send data to device

    // launch reverse() kernel
    reverse<<< 1, 5 >>>(dev_in, dev_out); //1 block of size threads

    // copy device result back to host copy of c
   hipMemcpy( d_out, dev_out, size,   hipMemcpyDeviceToHost );

   for(i=0;i<5;i++){
     printf(" %d ",d_in[i]);
   }
   printf("\n");
           
   for(i=0;i<5;i++){
     printf(" %d ",d_out[i]);
   }
   printf("\n");	

    free( d_out );
    hipFree( dev_in );
    hipFree( dev_out );
    return 0;
}
