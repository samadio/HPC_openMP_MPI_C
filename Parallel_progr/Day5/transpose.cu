#include "hip/hip_runtime.h"
#include <stdio.h>
#define row 4
#define col 5
#define space row*col*sizeof(int)
#define elements row*col

__global__ void tranpose (int* A, int*B){
  int row= gridDim.x;
  int col= blockDim.x;
  B[col+row*i]=A[row+col*i];
}


int main() {
    
  int*A=(int*)malloc(space);
  int* dev_A;
  int*B=(int*)malloc(space);
  int* dev_B;

  int i;
  for(i=0;i<elements;i++){
    A[i]=i;
  }

  // allocate device copies of A and B
  hipMalloc( (void*)&dev_A, space );
  hipMalloc( (void*)&dev_B, space );

  hipMemcpy( dev_A, A, size, hipMemcpyHostToDevice ); //send data to device

  // launch reverse() kernel
  transpose<<< row, col >>>(dev_A, dev_B);

  // copy device result back to host copy of c
  hipMemcpy( B, dev_B, size,   hipMemcpyDeviceToHost );

  for(i=0;i<elements;i++){
    if(i%col==0 && i!=0)printf("\n");
    printf("%d ", B[i]);
  }

  free(A); free(B);
  hipFree( dev_A ); hipFree( dev_B );
  return 0;
}
