
#include <hip/hip_runtime.h>
#include <stdio.h>
#define row 4
#define col 5
#define space 4*5*sizeof(int)
#define elements 4*5

__global__ void transpose (int* A, int*B){
  int rows=blockIdx.x;
  int column=threadIdx.x;
  int col_dim=blockDim.x;
  int row_dim=gridDim.x;
  B[column+rows*row_dim]=A[rows+column*col_dim];
}


int main() {
    
  int*A=(int*)malloc(space);
  int* dev_A;
  int*B=(int*)malloc(space);
  int* dev_B;

  int i;
  for(i=0;i<elements;i++){
    A[i]=i;
  }

  // allocate device copies of A and B
  hipMalloc( (void**)&dev_A, space );
  hipMalloc( (void**)&dev_B, space );

  hipMemcpy( dev_A, A, space, hipMemcpyHostToDevice ); //send data to device

  // launch transpose() kernel
  transpose<<< row, col >>>(dev_A, dev_B);

  // copy device result back to host copy of c
  hipMemcpy( B, dev_B, space,   hipMemcpyDeviceToHost );

  for(i=0;i<elements;i++){
    if(i%col==0 && i!=0)printf("\n");
      printf("%d ", A[i]);
  }
  printf("\n");

  for(i=0;i<elements;i++){
    if(i%row==0 && i!=0)printf("\n");
    printf("%d ", B[i]);
  }
  printf("\n");

  free(A); free(B);
  hipFree( dev_A ); hipFree( dev_B );
  return 0;
}
