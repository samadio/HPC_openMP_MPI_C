
#include <hip/hip_runtime.h>
#include <stdio.h>
#define row 4
#define col 5
#define space 4*5*sizeof(int)
#define elements 4*5

//this works until col<=Nthreads. In this case every threads moves only 1 data
__global__ void transpose (int* A, int*B){
  int i=threadIdx.x+(blockIdx.x*blockDim.x);
  int totlength= blockDim.x*gridDim.x-1;
  int factor;
  if(blockIdx.x==0) factor=1;
  else factor=blockIdx.x;
  int j= i*(gridDim.x) %(totlength*factor);
  B[j]=A[i];
}


/*x=threadidx, y=blockidx
* while (x<N){        //in case col>Nthreads
  M_out(y*N+x)=M_in(x*N+y);
} x+=blockdim.x
*
*/

int main() {
    
  int*A=(int*)malloc(space);
  int* dev_A;
  int*B=(int*)malloc(space);
  int* dev_B;

  int i;
  for(i=0;i<elements;i++){
    A[i]=i;
  }

  // allocate device copies of A and B
  hipMalloc( (void**)&dev_A, space );
  hipMalloc( (void**)&dev_B, space );

  hipMemcpy( dev_A, A, space, hipMemcpyHostToDevice ); //send data to device

  // launch transpose() kernel
  transpose<<< row, col >>>(dev_A, dev_B); 

  // copy device result back to host copy of c
  hipMemcpy( B, dev_B, space,   hipMemcpyDeviceToHost );

  for(i=0;i<elements;i++){
    if(i%col==0 && i!=0)printf("\n");
      printf("%d ", A[i]);
  }
  printf("\n");

  for(i=0;i<elements;i++){
    if(i%row==0 && i!=0)printf("\n");
    printf("%d ", B[i]);
  }
  printf("\n");

  free(A); free(B);
  hipFree( dev_A ); hipFree( dev_B );
  return 0;
}
