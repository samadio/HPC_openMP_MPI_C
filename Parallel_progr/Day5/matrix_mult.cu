
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 2048
#define nthreads 512


__global__ void matrix_mul (int* A, int*B,int*C,int size){
  int i=threadIdx.x+(blockIdx.x*blockDim.x);
  int rowidx= (i/size)*size;
  int colidx= i%size;
  int acc=0;
  int k;
  
  for(k=0; k<size;k++){
    acc+=A[rowidx+k]*B[colidx+k*size];
  }
  C[i]=acc;
}


int main() {
    
  int*A=(int*)malloc(N*N*sizeof(int));
  int* dev_A;
  int*B=(int*)malloc(N*N*sizeof(int));
  int* dev_B;
  int*C=(int*)malloc(N*N*sizeof(int));
  int* dev_C;

  int i;
  for(i=0;i<N*N;i++){
    A[i]=i;
  }
  
  for(i=0;i<N*N;i++){
    B[i]=i;
  }


  // allocate space for all the three matrixes
  hipMalloc( (void**)&dev_A,  N*N*sizeof(int));
  hipMalloc( (void**)&dev_B, N*N*sizeof(int) );
  hipMalloc( (void**)&dev_C, N*N*sizeof(int) );

  //send data to device
  hipMemcpy( dev_A, A, N*N*sizeof(int), hipMemcpyHostToDevice );
  hipMemcpy( dev_B, B, N*N*sizeof(int), hipMemcpyHostToDevice );

  // launch matrix_mul kernel
  matrix_mul<<< (N*N)/nthreads, nthreads >>>(dev_A, dev_B, dev_C, N);

  // copy results
  hipMemcpy( C, dev_C, N*N*sizeof(int),   hipMemcpyDeviceToHost );

/*  for(i=0;i<N*N;i++){
    if(i%N==0 && i!=0)printf("\n");
      printf("%d ", C[i]);
  }
  printf("\n"); */

  free(A); free(B);free(C);
  hipFree( dev_A ); hipFree( dev_B );hipFree( dev_C );
  return 0;
}
